#include <iostream>
#include "agent/Agent.h"
#include "obstacle/Obstacle.h"
#include "workspace/Workspace.h"
#include "agent/Agent.h"
#include "planners/RRT.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdio>
#include <Eigen/Core>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/functional.h>

int main(void){

    const int N = 6;
    int h_data[N] = {1, 2, 3, 4, 5, 6};

    // Transfer data to the device
    thrust::device_vector<int> d_data(h_data, h_data + N);

    // Apply a transformation (multiply each element by 2)
    thrust::transform(d_data.begin(), d_data.end(), d_data.begin(), thrust::placeholders::_1 * 2);

    // Transfer data back to the host
    thrust::copy(d_data.begin(), d_data.end(), h_data);

    // Print the results
    std::cout << "Result: ";
    for (int i = 0; i < N; i++) {
        std::cout << h_data[i] << " ";
    }
    std::cout << std::endl;

    printf("TEST2\n");
    return 0;
}