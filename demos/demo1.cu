#include <iostream>
#include "agent/Agent.h"
#include "obstacle/Obstacle.h"
#include "workspace/Workspace.h"
#include "agent/Agent.h"
#include "planners/RRT.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdio>
#include <Eigen/Core>


int main(void){

    // Obstacle o = Obstacle();
    // o.verticesCCW.push_back(Eigen::Vector2d(0, 0));
    // o.verticesCCW.push_back(Eigen::Vector2d(1, 0));
    // o.verticesCCW.push_back(Eigen::Vector2d(1, 1));
    // o.verticesCCW.push_back(Eigen::Vector2d(0, 1));
    // std::cout<<"TEST"<<std::endl;
    Workspace w = Workspace(10,10);
    Obstacle o1 = Obstacle(5,5,1);
    Obstacle o2 = Obstacle(2,2,1);
    Obstacle o3 = Obstacle(8,8,1);
    w.obstacles_.push_back(o1);
    w.obstacles_.push_back(o2);
    w.obstacles_.push_back(o3);

    Agent a = Agent(1,1,.01);

    RRT r = RRT();
    r.plan(w,a,Eigen::Vector2d(9,9));

    printf("TEST2\n");
    return 0;
}