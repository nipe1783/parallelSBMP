#include "hip/hip_runtime.h"
#include "planners/Planner.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdio>
#include <Eigen/Core>
#include "obstacle/Obstacle.h"
#include "workspace/Workspace.h"
#include "agent/Agent.h"
#include "state/State.cuh"

// Global: called from the CPU to the GPU
__global__ void generateRandomSampleKernel(State* samples, int numSamples){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < numSamples) {
        samples[idx] = Planner::generateRandomSampleDevice();
    }
}

// Host function to manage GPU memory and launch kernel
std::vector<State> Planner::generateRandomSamples(const State &state, int numSamples) {
    State* d_samples; // Device memory pointer (GPU)
    std::vector<State> h_samples(numSamples); // Host memory vector (CPU)

    // Allocate device memory
    hipMalloc(&d_samples, numSamples * sizeof(State)); // Allocate memory on the GPU

    // Launch kernel
    int threadsPerBlock = 256; // Number of threads per block
    int blocksPerGrid = (numSamples + threadsPerBlock - 1) / threadsPerBlock; // Number of blocks per grid
    generateRandomSampleKernel<<<blocksPerGrid, threadsPerBlock>>>(d_samples, numSamples); // Launch kernel
    hipDeviceSynchronize(); // Wait for the GPU to finish

    // Copy result back to host
    hipMemcpy(h_samples.data(), d_samples, numSamples * sizeof(Eigen::Vector2d), hipMemcpyDeviceToHost); // Copy memory from GPU to CPU

    // Free device memory
    hipFree(d_samples); // Free memory on the GPU

    return h_samples; // Return the samples
}

// Device: called from the GPU to the GPU
__device__ State Planner::generateRandomSampleDevice(){
    return State(0.0f, 0.0f);
}

