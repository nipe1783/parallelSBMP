#include "hip/hip_runtime.h"
#include "planners/Planner.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdio>
#include <Eigen/Core>
#include "obstacle/Obstacle.h"
#include "workspace/Workspace.h"
#include "agent/Agent.h"

// Global: called from the CPU to the GPU
__global__ void generateRandomSampleKernel(Eigen::Vector2d* samples, int numSamples){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < numSamples) {
        samples[idx] = Planner::generateRandomSampleDevice();
    }
}

// Host function to manage GPU memory and launch kernel
std::vector<Eigen::Vector2d> Planner::generateRandomSamples() {
    const int numSamples = 100; // Number of samples to generate
    Eigen::Vector2d* d_samples; // Device memory pointer (GPU)
    std::vector<Eigen::Vector2d> h_samples(numSamples); // Host memory vector (CPU)

    // Allocate device memory
    hipMalloc(&d_samples, numSamples * sizeof(Eigen::Vector2d)); // Allocate memory on the GPU

    // Launch kernel
    int threadsPerBlock = 256; // Number of threads per block
    int blocksPerGrid = (numSamples + threadsPerBlock - 1) / threadsPerBlock; // Number of blocks per grid
    generateRandomSampleKernel<<<blocksPerGrid, threadsPerBlock>>>(d_samples, numSamples); // Launch kernel
    hipDeviceSynchronize(); // Wait for the GPU to finish

    // Copy result back to host
    hipMemcpy(h_samples.data(), d_samples, numSamples * sizeof(Eigen::Vector2d), hipMemcpyDeviceToHost); // Copy memory from GPU to CPU

    // Free device memory
    hipFree(d_samples); // Free memory on the GPU

    return h_samples; // Return the samples
}

// Device: called from the GPU to the GPU
__device__ Eigen::Vector2d Planner::generateRandomSampleDevice(){
    return Eigen::Vector2d(0, 0);
}

