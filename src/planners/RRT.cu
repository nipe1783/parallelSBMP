#include "planners/RRT.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdio>
#include <Eigen/Core>
#include "obstacle/Obstacle.h"
#include "workspace/Workspace.h"
#include "agent/Agent.h"

void RRT::plan(const Workspace& workspace, const Agent& agent, const Eigen::Vector2d& goal){
    int numSamples = 100;
    State state(0.0f, 0.0f);
    for (int i = 0; i < 100; i++){
        std::vector<State> samples = generateRandomSamples(state, numSamples);
        printf("Number of samples: %d\n", samples.size());
    }
    
}