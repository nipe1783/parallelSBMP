#include "planners/RRT.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdio>
#include <Eigen/Core>
#include "obstacle/Obstacle.h"
#include "workspace/Workspace.h"
#include "agent/Agent.h"

void RRT::plan(const Workspace& workspace, const Agent& agent, const Eigen::Vector2d& goal){
    for (int i = 0; i < 100; i++){
        std::vector<Eigen::Vector2d> samples = generateRandomSamples();
        printf("Number of samples: %d\n", samples.size());
    }
    
}