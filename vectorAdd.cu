#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdio>

__global__ void vectorAdd(const float *A, const float *B, float *C) {
    int i = threadIdx.x;
    printf("Thread %d: A[%d] = %f, B[%d] = %f\n", i, i, A[i], i, B[i]);
    C[i] = A[i] + B[i];
}

int main(void){
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElements = 50000;
    size_t size = numElements * sizeof(float);
    printf("[Vector addition of %d elements]\n", numElements);

    // Allocate the host input vector A
    float *h_A = (float *)malloc(size);

    // Allocate the host input vector B
    float *h_B = (float *)malloc(size);

    // Allocate the host output vector C
    float *h_C = (float *)malloc(size);

    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL) {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i) {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
    }
}